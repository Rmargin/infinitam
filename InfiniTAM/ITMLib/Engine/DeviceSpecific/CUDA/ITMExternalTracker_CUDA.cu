#include "hip/hip_runtime.h"
// Copyright 2014-2015 Isis Innovation Limited and the authors of InfiniTAM

#include "../../DeviceAgnostic/ITMExternalTracker.h"
#include "../../DeviceAgnostic/ITMPixelUtils.h"
#include "ITMCUDAUtils.h"
#include "ITMExternalTracker_CUDA.h"

using namespace ITMLib::Engine;

//__global__ void externalTrackerOneLevel_f_device(
//    Vector2f* out, Vector4f* locations, Vector4f* colours, Vector4u* rgb,
//    int noTotalPoints, Matrix4f M, Vector4f projParams, Vector2i imgSize);
//
//__global__ void externalTrackerOneLevel_g_rt_device(
//    float* g_out, float* h_out, Vector4f* locations, Vector4f* colours,
//    Vector4s* gx, Vector4s* gy, Vector4u* rgb, int noTotalPoints, Matrix4f M,
//    Vector4f projParams, Vector2i imgSize);
//__global__ void externalTrackerOneLevel_g_ro_device(
//    float* g_out, float* h_out, Vector4f* locations, Vector4f* colours,
//    Vector4s* gx, Vector4s* gy, Vector4u* rgb, int noTotalPoints, Matrix4f M,
//    Vector4f projParams, Vector2i imgSize);

// host methods

ITMExternalTracker_CUDA::ITMExternalTracker_CUDA(
    Vector2i imgSize, TrackerIterationType* trackingRegime,
    int noHierarchyLevels, const ITMLowLevelEngine* lowLevelEngine)
    : ITMExternalTracker(imgSize, trackingRegime, noHierarchyLevels,
                         lowLevelEngine, MEMORYDEVICE_CUDA) {
//  int dim_g = 6;
//  int dim_h = 6 + 5 + 4 + 3 + 2 + 1;
//
//  ITMSafeCall(hipMalloc((void**)&f_device,
//                         sizeof(Vector2f) * (imgSize.x * imgSize.y / 128)));
//  ITMSafeCall(hipMalloc((void**)&g_device, sizeof(float) * dim_g *
//                                                (imgSize.x * imgSize.y / 128)));
//  ITMSafeCall(hipMalloc((void**)&h_device, sizeof(float) * dim_h *
//                                                (imgSize.x * imgSize.y / 128)));
//
//  f_host = new Vector2f[imgSize.x * imgSize.y / 128];
//  g_host = new float[dim_g * imgSize.x * imgSize.y / 128];
//  h_host = new float[dim_h * imgSize.x * imgSize.y / 128];
}

ITMExternalTracker_CUDA::~ITMExternalTracker_CUDA(void) {
//  ITMSafeCall(hipFree(f_device));
//  ITMSafeCall(hipFree(g_device));
//  ITMSafeCall(hipFree(h_device));
//
//  delete[] f_host;
//  delete[] g_host;
//  delete[] h_host;
}

void ITMExternalTracker_CUDA::F_oneLevel(float* f, ITMPose* pose) {
//  int noTotalPoints = trackingState->pointCloud->noTotalPoints;
//
//  Vector4f projParams = view->calib->intrinsics_rgb.projectionParamsSimple.all;
//  projParams.x /= 1 << levelId;
//  projParams.y /= 1 << levelId;
//  projParams.z /= 1 << levelId;
//  projParams.w /= 1 << levelId;
//
//  Matrix4f M = pose->GetM();
//
//  Vector2i imgSize = viewHierarchy->levels[levelId]->rgb->noDims;
//
//  float scaleForOcclusions, final_f;
//
//  Vector4f* locations =
//      trackingState->pointCloud->locations->GetData(MEMORYDEVICE_CUDA);
//  Vector4f* colours =
//      trackingState->pointCloud->colours->GetData(MEMORYDEVICE_CUDA);
//  Vector4u* rgb =
//      viewHierarchy->levels[levelId]->rgb->GetData(MEMORYDEVICE_CUDA);
//
//  dim3 blockSize(128, 1);
//  dim3 gridSize((int)ceil((float)noTotalPoints / (float)blockSize.x), 1);
//
//  ITMSafeCall(hipMemset(f_device, 0, sizeof(Vector2f) * gridSize.x));
//
//  externalTrackerOneLevel_f_device<<<gridSize, blockSize>>>(
//      f_device, locations, colours, rgb, noTotalPoints, M, projParams, imgSize);
//
//  ITMSafeCall(hipMemcpy(f_host, f_device, sizeof(Vector2f) * gridSize.x,
//                         hipMemcpyDeviceToHost));
//
//  final_f = 0;
//  countedPoints_valid = 0;
//  for (size_t i = 0; i < gridSize.x; i++) {
//    final_f += f_host[i].x;
//    countedPoints_valid += (int)f_host[i].y;
//  }
//
//  if (countedPoints_valid == 0) {
//    final_f = MY_INF;
//    scaleForOcclusions = 1.0;
//  } else {
//    scaleForOcclusions = (float)noTotalPoints / countedPoints_valid;
//  }
//
//  f[0] = final_f * scaleForOcclusions;
}

void ITMExternalTracker_CUDA::G_oneLevel(float* gradient, float* hessian,
                                         ITMPose* pose) const {
//  int noTotalPoints = trackingState->pointCloud->noTotalPoints;
//
//  Vector4f projParams = view->calib->intrinsics_rgb.projectionParamsSimple.all;
//  projParams.x /= 1 << levelId;
//  projParams.y /= 1 << levelId;
//  projParams.z /= 1 << levelId;
//  projParams.w /= 1 << levelId;
//
//  Matrix4f M = pose->GetM();
//
//  Vector2i imgSize = viewHierarchy->levels[levelId]->rgb->noDims;
//
//  float scaleForOcclusions;
//
//  bool rotationOnly = iterationType == TRACKER_ITERATION_ROTATION;
//  int numPara = rotationOnly ? 3 : 6,
//      numParaSQ = rotationOnly ? 3 + 2 + 1 : 6 + 5 + 4 + 3 + 2 + 1;
//
//  float globalGradient[6], globalHessian[21];
//  for (int i = 0; i < numPara; i++) globalGradient[i] = 0.0f;
//  for (int i = 0; i < numParaSQ; i++) globalHessian[i] = 0.0f;
//
//  Vector4f* locations =
//      trackingState->pointCloud->locations->GetData(MEMORYDEVICE_CUDA);
//  Vector4f* colours =
//      trackingState->pointCloud->colours->GetData(MEMORYDEVICE_CUDA);
//  Vector4u* rgb =
//      viewHierarchy->levels[levelId]->rgb->GetData(MEMORYDEVICE_CUDA);
//  Vector4s* gx =
//      viewHierarchy->levels[levelId]->gradientX_rgb->GetData(MEMORYDEVICE_CUDA);
//  Vector4s* gy =
//      viewHierarchy->levels[levelId]->gradientY_rgb->GetData(MEMORYDEVICE_CUDA);
//
//  dim3 blockSize(128, 1);
//  dim3 gridSize((int)ceil((float)noTotalPoints / (float)blockSize.x), 1);
//
//  if (rotationOnly) {
//    ITMSafeCall(hipMemset(g_device, 0, sizeof(float) * gridSize.x * 3));
//    ITMSafeCall(hipMemset(h_device, 0, sizeof(float) * gridSize.x * 6));
//
//    externalTrackerOneLevel_g_ro_device<<<gridSize, blockSize>>>(
//        g_device, h_device, locations, colours, gx, gy, rgb, noTotalPoints, M,
//        projParams, imgSize);
//  } else {
//    ITMSafeCall(hipMemset(g_device, 0, sizeof(float) * gridSize.x * 6));
//    ITMSafeCall(hipMemset(h_device, 0, sizeof(float) * gridSize.x * 21));
//
//    externalTrackerOneLevel_g_rt_device<<<gridSize, blockSize>>>(
//        g_device, h_device, locations, colours, gx, gy, rgb, noTotalPoints, M,
//        projParams, imgSize);
//  }
//
//  ITMSafeCall(hipMemcpy(g_host, g_device, sizeof(float) * gridSize.x * numPara,
//                         hipMemcpyDeviceToHost));
//  ITMSafeCall(hipMemcpy(h_host, h_device,
//                         sizeof(float) * gridSize.x * numParaSQ,
//                         hipMemcpyDeviceToHost));
//
//  for (size_t i = 0; i < gridSize.x; i++) {
//    for (int p = 0; p < numPara; p++)
//      globalGradient[p] += g_host[i * numPara + p];
//    for (int p = 0; p < numParaSQ; p++)
//      globalHessian[p] += h_host[i * numParaSQ + p];
//  }
//
//  scaleForOcclusions = (float)noTotalPoints / countedPoints_valid;
//  if (countedPoints_valid == 0) {
//    scaleForOcclusions = 1.0f;
//  }
//
//  for (int para = 0, counter = 0; para < numPara; para++) {
//    gradient[para] = globalGradient[para] * scaleForOcclusions;
//    for (int col = 0; col <= para; col++, counter++)
//      hessian[para + col * numPara] =
//          globalHessian[counter] * scaleForOcclusions;
//  }
//  for (int row = 0; row < numPara; row++) {
//    for (int col = row + 1; col < numPara; col++)
//      hessian[row + col * numPara] = hessian[col + row * numPara];
//  }
}
