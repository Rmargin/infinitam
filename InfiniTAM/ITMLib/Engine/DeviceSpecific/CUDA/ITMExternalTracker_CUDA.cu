#include "hip/hip_runtime.h"
// Copyright 2014-2015 Isis Innovation Limited and the authors of InfiniTAM

#include "../../DeviceAgnostic/ITMExternalTracker.h"
#include "../../DeviceAgnostic/ITMPixelUtils.h"
#include "ITMCUDAUtils.h"
#include "ITMExternalTracker_CUDA.h"
#include <glog/logging.h>

using namespace ITMLib::Engine;

//__global__ void externalTrackerOneLevel_f_device(
//    Vector2f* out, Vector4f* locations, Vector4f* colours, Vector4u* rgb,
//    int noTotalPoints, Matrix4f M, Vector4f projParams, Vector2i imgSize);
//
//__global__ void externalTrackerOneLevel_g_rt_device(
//    float* g_out, float* h_out, Vector4f* locations, Vector4f* colours,
//    Vector4s* gx, Vector4s* gy, Vector4u* rgb, int noTotalPoints, Matrix4f M,
//    Vector4f projParams, Vector2i imgSize);
//__global__ void externalTrackerOneLevel_g_ro_device(
//    float* g_out, float* h_out, Vector4f* locations, Vector4f* colours,
//    Vector4s* gx, Vector4s* gy, Vector4u* rgb, int noTotalPoints, Matrix4f M,
//    Vector4f projParams, Vector2i imgSize);

// host methods

ITMExternalTracker_CUDA::ITMExternalTracker_CUDA(
    Vector2i imgSize, TrackerIterationType* trackingRegime,
    int noHierarchyLevels, const ITMLowLevelEngine* lowLevelEngine)
    : ITMExternalTracker(imgSize, trackingRegime, noHierarchyLevels,
                         lowLevelEngine, MEMORYDEVICE_CUDA) {
  LOG(ERROR) << "not implemented yet";
}

ITMExternalTracker_CUDA::~ITMExternalTracker_CUDA(void) {
  LOG(ERROR) << "not implemented yet";
}

void ITMExternalTracker_CUDA::F_oneLevel(float* f, ITMPose* pose) {
  LOG(ERROR) << "not implemented yet";
}

void ITMExternalTracker_CUDA::G_oneLevel(float* gradient, float* hessian,
                                         ITMPose* pose) const {
  LOG(ERROR) << "not implemented yet";
}
